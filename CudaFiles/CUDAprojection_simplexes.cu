
#include <hip/hip_runtime.h>
__device__ void CUDAprojection_simplexes_sort_quickSort_partition(int *pi, double *x, int low, int high, int t, int T)
{
    /* pivot (Element to be placed at right position) */
    double pivot = x[high*T + t];
 
    int i = (low - 1);  /* Index of smaller element */
	double swap;

    for(int j = low; j <= high - 1; j++)
    {
        // If current element is smaller than or
        // equal to pivot
        if(x[j*T + t] <= pivot)
        {
            i++;    // increment index of smaller element
            
			swap = x[i*T+t];
			x[i*T+t] = x[j*T+t];
			x[j*T+t] = swap;
        }
    }

    swap = x[(i+1)*T+t];
	x[(i+1)*T+t] = x[high*T+t];
	x[high*T+t] = swap;

    *pi = i + 1;
}

__device__ void CUDAprojection_simplexes_sort_quickSort(double *x, int low, int high, int t, int T)
{
    if (low < high)
    {
        /* pi is partitioning index, arr[pi] is now
           at right place */
		int pi;
		CUDAprojection_simplexes_sort_quickSort_partition(&pi, x, low, high, t,T);

		CUDAprojection_simplexes_sort_quickSort(x, low, pi - 1, t,T);  /* Before pi */
		CUDAprojection_simplexes_sort_quickSort(x, pi + 1, high, t,T); /* After pi */
    }
}

__device__ void device_sort_bubble(double *x_sorted, int t, int T, int K){
	int i;
	int m=K;
	int mnew;
	double swap;

	while(m > 0){
		/* Iterate through x */
		mnew = 0;
		for(i=1;i<m;i++){
			/* Swap elements in wrong order */
			if (x_sorted[i*T+t] < x_sorted[(i - 1)*T + t]){
				swap = x_sorted[i*T + t];
				x_sorted[i*T + t] = x_sorted[(i - 1)*T + t];
				x_sorted[(i - 1)*T + t] = swap;
				mnew = i;
			}
        }
		m = mnew;
	}
}

__global__ void CUDAprojection_simplexes( double *X, 
                   double *Y,
                   int T, int K ) {

	int t = blockIdx.x*blockDim.x + threadIdx.x;
	
	int k;

	if(t<T){
		bool is_inside = true;
		double sum = 0.0;
	
		/* control inequality constraints */
		for(k = 0; k < K; k++){ // TODO: could be performed parallely  
			if(X[k*T+t] < 0.0){
				is_inside = false;
			}
			sum += X[k*T + t];
			
			Y[k*T + t] = X[k*T + t];
		}

		/* control equality constraints */
		if(sum != 1){ 
			is_inside = false;
		}

		/* if given point is not inside the feasible domain, then do projection */
		if(!is_inside){
			int j,i;
			/* compute sorted x_sub */
			double sum_y;

//			CUDAprojection_simplexes_sort_bubble(Y,t,T,K);
            CUDAprojection_simplexes_sort_quickSort(Y, 0, K-1, t, T);

			/* now perform analytical solution of projection problem */	
			double t_hat = 0.0;
			i = K - 1;
			double ti;

			while(i >= 1){
				/* compute sum(y) */
				sum_y = 0.0;
				for(j=i;j<K;j++){ /* sum(y(i,n-1)) */
					sum_y += Y[j*T + t];
				}
				
				ti = (sum_y - 1.0)/(double)(K-i);
				if(ti >= Y[(i-1)*T + t]){
					t_hat = ti;
					i = -1; /* break */
				} else {
					i = i - 1;
				}
			}

			if(i == 0){
				t_hat = (sum-1.0)/(double)K; /* uses sum=sum(x_sub) */
			}
    
			for(k = 0; k < K; k++){ // TODO: could be performed parallely  
				/* (*x_sub)(i) = max(*x_sub-t_hat,0); */
				ti = X[k*T + t] - t_hat;	
				if(ti > 0.0){
					X[k*T + t] = ti;
				} else {
					X[k*T + t] = 0.0;
				}
			}
		}
		
	}

	/* if t >= T then relax and do nothing */
}
